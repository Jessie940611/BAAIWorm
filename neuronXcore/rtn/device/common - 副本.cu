#include "hip/hip_runtime.h"
﻿// ======================================================================== //
// Copyright 2018-2020 The Contributors                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "rtn/device/TubesGeom.h"
#include "rtn/device/PerRayData.h"
#include "rtn/device/RayGenData.h"
#include "rtn/device/Camera.h"
#include "rtn/device/disney_bsdf.h"

#include <optix_device.h> // Only for test, can be remove later. 

#define HIGH -24.0f//37.1453f//-24.0f
#define MID1 -42.0f
#define MID2 -60.0f
#define LOW  -78.0f//-90.2629f//-100.0f
#define DELTA 18.0f

//:37.1453 Min : -90.2629

namespace rtn {
	namespace device {


		inline __device__ vec3f short_rainbow_rgb(float v)
		{
			v = clamp(v,LOW, HIGH);
			v = (v - LOW) / (HIGH - LOW);

			float a = (1.f - v) / 0.25f;	//invert and group
			int X = floor(a);	//this is the integer part
			int Y = floor(255 * (a - X)); //fractional part from 0 to 255
			int r, g, b;

			switch (X)
			{
				case 0: r = 255; g = Y; b = 0; break;
				case 1: r = 255 - Y; g = 255; b = 0; break;
				case 2: r = 0; g = 255; b = Y; break;
				case 3: r = 0; g = 255 - Y; b = 255; break;
				case 4: r = 0; g = 0; b = 255; break;
			}
			return vec3f(r/255.f, g/255.f, b/255.f);
	    }

		// Lei : change this for a better vol --> color.
		inline __device__ vec3f vol2rgb(float v) {
			float red, green, blue;
			if (v > HIGH)
			{
				red = green = 1.0f;
				blue = (v - HIGH) / DELTA;
				if (blue > 1.0f)
				{
					blue = 1.0f;
				}
			}
			else if (v > MID1)
			{
				red = 1.0f;
				green = (v - MID1) / DELTA;
				blue = 0.0f;
			}
			else if (v > MID2)
			{
				red = (v - MID2) / DELTA;
				green = (MID1 - v) / (2 * DELTA);
				blue = 0.0f;
			}
			else if (v > LOW)
			{
				red = 0.0f;
				green = (v - LOW) / (2 * DELTA);
				blue = (MID2 - v) / DELTA;
			}
			else
			{
				red = green = 0.0f;
				blue = 1.0 - (LOW - v) / DELTA;
				if (blue < 0.0f)
				{
					blue = 0.0f;
				}
			}
			return vec3f(red, green, blue);
		}

		inline __device__
			int32_t make_8bit(const float f)
		{
			return min(255, max(0, int(f * 256.f)));
		}

		inline __device__
			int32_t make_rgba8(const vec4f color)
		{
			return
				(make_8bit(color.x) << 0) +
				(make_8bit(color.y) << 8) +
				(make_8bit(color.z) << 16);
		}

		inline __device__ vec3f random_in_unit_sphere(Random& rnd) {
			vec3f p;
			do {
				p = 2.0f * vec3f(rnd(), rnd(), rnd()) - vec3f(1.f, 1.f, 1.f);
			} while (dot(p, p) >= 1.0f);
			return p;
		}

		// ------------------------------------------------------------------
		// actual tubes stuff
		// ------------------------------------------------------------------

		inline __device__
			vec3f missColor(const Ray& ray, vec3f top_color, vec3f bottom_color)
		{
			const vec2i pixelID = owl::getLaunchIndex();
			const float t = pixelID.y / (float)optixGetLaunchDimensions().y;
			const vec3f c = (1.0f - t) * top_color + t * bottom_color;
			return c;
		}


		inline __device__
			vec3f pathTrace(const RayGenData& self,
				owl::Ray& ray,
				Random& rnd,
				PerRayData& prd)
		{
			vec3f attenuation = 1.f;

			const FrameState* fs = &self.frameStateBuffer[0];
			int pathDepth = fs->pathDepth;

			vec3f ambientLight(fs->ambient /*1.8f*/);

			/* ambientLight *= fs->frameID / 60;*/

			 /* code for tubes */
			if (pathDepth <= 1)
			{
				prd.primID = -1;
				owl::traceRay(/*accel to trace against*/self.world,
					/*the ray to trace*/ ray,
					/*prd*/prd,
					OPTIX_RAY_FLAG_DISABLE_ANYHIT);

				if (prd.primID < 0)
					return missColor(ray, vec3f(0.5f), vec3f(fs->bg));

				vec3f N = prd.Ng;
				if (dot(N, (vec3f)ray.direction) > 0.f)
					N = -N;
				N = normalize(N);

				// hardcoded albedo for now:
				vec3f albedo;

				// Tractography colors
				if (fs->shadeMode == 1 && prd.meshID < 0)
				{
					unsigned colorID = self.primColors[prd.primID];
					colorID = clamp(colorID, (unsigned)0, (unsigned)12085);
					float rgba = self.voltagesColor[colorID];
					float val = rgba ;
					albedo = vol2rgb(val);
					//albedo = short_rainbow_rgb(val);
					// move this color stuff to CH program!?
					// TODO: this assumes that primitives are lines!!
					//vec3f pa = self.vertices[prd.primID*2];
					//vec3f pb = self.vertices[prd.primID*2+1];
					 //albedo = abs(normalize(pa - pb)); //vec3f(0.628, 0.85, 0.511);
				}
				else
				{
					// Random colors
					albedo = randomColor(1 + prd.primID);
				}
				vec3f color = albedo * (.2f + .6f * fabsf(dot(N, (vec3f)ray.direction)));
				return color;
			}

			// could actually swtich material based on meshID ...
			DisneyMaterial material = fs->material;
			/* iterative version of recursion, up to depth 50 */
			for (int depth = 0; true; depth++)
			{
				prd.primID = -1;
				owl::traceRay(/*accel to trace against*/self.world,
					/*the ray to trace*/ ray,
					/*prd*/prd,
					OPTIX_RAY_FLAG_DISABLE_ANYHIT);

				if (prd.primID == -1)
				{
					// miss...
					if (depth == 0)
						return missColor(ray, vec3f(0.06f, 0.11f, 0.26f), vec3f(fs->bg));

#if FAST_SHADING
					return attenuation * ambientLight;
#else
					float phi = atan2(ray.direction.z, ray.direction.x); // azimuth 
					float theta = acos(ray.direction.y / length(ray.direction)); // elevation
					const float half_width = M_PIF / 8.f;
					const float elevation = M_PIF / 4.f; // from 0 to PI
					const float azimuth = M_PIF / 2.f;  // from 0 to 2PI
					if (theta > (elevation - half_width) && theta < (elevation + half_width)
						&& phi >(azimuth - half_width) && phi < (azimuth + half_width)) {
						return attenuation * owl::vec3f(fs->light);
					}
					else {
						return attenuation * owl::vec3f(ambientLight / 2.f);
					}
#endif
				}

				vec3f N = normalize(prd.Ng);
				const vec3f w_o = -ray.direction;
				if (dot(N, w_o) < 0.f)
				{
					N = -N;
				}

				if (prd.meshID >= 0)
				{
					/* kinda hacky, check if mesh has vertex colors. if not, use material base color. */
					if (!(prd.color.x < 0))
					{
						material.base_color = prd.color;
					}
				}
				else
				{
					// Tractography colors
					if (fs->shadeMode == 1)
					{
						// TODO: this assumes that primitives are lines!!
						//vec3f pa = self.vertices[prd.primID*2];
						//vec3f pb = self.vertices[prd.primID*2+1];
						//material.base_color = abs(normalize(pa - pb)); //vec3f(0.628, 0.85, 0.511);
						unsigned colorID = self.primColors[prd.primID];
						colorID = clamp(colorID, (unsigned)0, (unsigned)12085);
						float rgba = self.voltagesColor[colorID];
						float val = rgba;
						material.base_color = vol2rgb(val);
						//material.base_color = short_rainbow_rgb(val);
						//material.base_color = vec3f((rgba & 0xff) / 255.f,
						//    ((rgba >> 8) & 0xff) / 255.f,
						//    ((rgba >> 16) & 0xff) / 255.f);
						
					}
					// Other shade modes here...
					else if (fs->shadeMode == 2)
					{
						if (self.primColors[prd.primID] != unsigned(-1))
						{
							unsigned rgba = self.primColors[prd.primID]; // ignore alpha for now
							material.base_color = vec3f((rgba & 0xff) / 255.f,
								((rgba >> 8) & 0xff) / 255.f,
								((rgba >> 16) & 0xff) / 255.f);
						}
						else
							material.base_color = vec3f(35.0f / 255.0f, 60.0f / 255.0f, 43.0f / 255.0f);
					}
				}

				owl::vec3f v_x, v_y;
				ortho_basis(v_x, v_y, N);
				// pdf and dir are set by sampling the BRDF
				float pdf;
				vec3f scattered_direction;
				vec3f albedo = sample_disney_brdf(material, N, w_o, v_x, v_y, rnd, scattered_direction, pdf);

				const vec3f scattered_origin = ray.origin + prd.t * ray.direction;
				ray = owl::Ray(/* origin   : */ scattered_origin,
					/* direction: */ scattered_direction,
					/* tmin     : */ 1e-3f,
					/* tmax     : */ 1e+8f);

				if (depth >= pathDepth || pdf == 0.f || albedo == owl::vec3f(0.f))
				{
					// ambient term:
					return owl::vec3f(0.f);//attenuation * ambientLight;
				}

				attenuation *= albedo * fabs(dot(scattered_direction, N)) / pdf;
			}
		}

		OPTIX_MISS_PROGRAM(miss_program)()
		{
			/*! nothing to do - we initialize prd before trace */
		}

		/*! the actual ray generation program - note this has no formal
		  function parameters, but gets its paramters throught the 'pixelID'
		  and 'pixelBuffer' variables/buffers declared above */
		OPTIX_RAYGEN_PROGRAM(raygen_program)()
		{
			const RayGenData& self = owl::getProgramData<RayGenData>();
			const vec2i pixelID = owl::getLaunchIndex();
			const vec2i launchDim = owl::getLaunchDims();

			if (pixelID.x >= self.fbSize.x) return;
			if (pixelID.y >= self.fbSize.y) return;
			const int pixelIdx = pixelID.x + self.fbSize.x * pixelID.y;

			// for multi-gpu: only render every deviceCount'th column of 32 pixels:
			if (((pixelID.x / 32) % self.deviceCount) != self.deviceIndex)
				return;

			uint64_t clock_begin = clock64();
			const FrameState* fs = &self.frameStateBuffer[0];
			int pixel_index = pixelID.y * launchDim.x + pixelID.x;
			vec4f col(0.f);
			Random rnd(pixel_index, fs->accumID );

			PerRayData prd;

			for (int s = 0; s < fs->samplesPerPixel; s++)
			{
				vec2f pixelSample = vec2f(pixelID) + vec2f(rnd(), rnd());
				float u = float(pixelID.x + rnd());
				float v = float(pixelID.y + rnd());
				owl::Ray ray = Camera::generateRay(*fs, pixelSample, rnd);
				col += vec4f(pathTrace(self, ray, rnd, prd), 1);
			}
			col = col / float(fs->samplesPerPixel);

			uint64_t clock_end = clock64();
			if (fs->heatMapEnabled)
			{
				float t = (clock_end - clock_begin) * fs->heatMapScale;
				if (t >= 256.f * 256.f * 256.f)
					col = vec4f(1, 0, 0, 1);
				else
				{
					uint64_t ti = uint64_t(t);
					col.x = ((ti >> 16) & 255) / 255.f;
					col.y = ((ti >> 8) & 255) / 255.f;
					col.z = ((ti >> 0) & 255) / 255.f;
				}
			}

			if (fs->accumID > 0)
				col = col + (vec4f)self.accumBufferPtr[pixelIdx];
			self.accumBufferPtr[pixelIdx] = col;

			uint32_t rgba = make_rgba8(col / (fs->accumID + 1.f));
			self.colorBufferPtr[pixelIdx] = rgba;
		}

	}
}

