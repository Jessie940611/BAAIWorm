#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2020 The Contributors                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "rtn/device/TubesGeom.h"
#include "rtn/device/PerRayData.h"
#include "rtn/device/RayGenData.h"

#include "rtn/device/TriangleMesh.h"
#include "rtn/device/Camera.h"
#include "rtn/device/roundedCone.h"

#include "rtn/device/disney_bsdf.h"

#include <optix_device.h> // Only for test, can be remove later. 

#define HIGH 10.0f
#define LOW -105.0f
#define DELTA 0.001f

#ifndef M_PIf
#define M_PIf       3.14159265358979323846f
#endif
namespace rtn {
	namespace device {

		__forceinline__ __device__ void cosine_sample_hemisphere(float u1, float u2, vec3f& p)
		{
			// Uniformly sample disk.
			const float r = sqrtf(u1);
			const float phi = 2.0f * M_PIf * u2;
			p.x = r * cosf(phi);
			p.y = r * sinf(phi);

			// Project up to hemisphere.
			p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
		}
		struct Onb
		{
			__forceinline__ __device__ Onb(const vec3f& normal)
			{
				m_normal = normal;

				if (fabs(m_normal.x) > fabs(m_normal.z))
				{
					m_binormal.x = -m_normal.y;
					m_binormal.y = m_normal.x;
					m_binormal.z = 0;
				}
				else
				{
					m_binormal.x = 0;
					m_binormal.y = -m_normal.z;
					m_binormal.z = m_normal.y;
				}

				m_binormal = normalize(m_binormal);
				m_tangent = cross(m_binormal, m_normal);
			}

			__forceinline__ __device__ void inverse_transform(vec3f& p) const
			{
				p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
			}

			vec3f m_tangent;
			vec3f m_binormal;
			vec3f m_normal;
		};

		static inline __device__ vec3f voltage_mapping(float voltage, float maxVol, float minVol, float inhibition, float resting, float excitation, float firing)
		{
			float span = maxVol - minVol;
			float r, ri, rr, re, rf;
			r = (voltage - minVol) / span;
			ri = (inhibition - minVol) / span;
			rr = (resting - minVol) / span;
			re = (excitation - minVol) / span;
			rf = (firing - minVol) / span;
			float rt, ir, ig, ib;

			if (r <= 0.0f + 0.00001f)
			{
				ir = 0.0f;
				ig = 0.0f;
				ib = 0.0f;
			}
			if (0.0f + 0.00001f < r && r <= ri)
			{
				rt = r / ri;
				ir = 0.0f;
				ig = 0.0f;
				ib = 0.0f + 1.0f * rt;
			}
			if (ri < r && r <= rr)
			{
				rt = (r - ri) / (rr - ri);
				ir = 0.0f + 0.1f * rt;
				ig = 0.0f + 0.1f * rt;
				ib = 1.0f - 0.9f * rt;
			}
			if (rr < r && r <= re)
			{
				rt = (r - rr) / (re - rr);
				ir = 0.1f + 0.9f * rt;
				ig = 0.1f - 0.1f * rt;
				ib = 0.1f - 0.1f * rt;
			}
			if (re < r && r <= rf)
			{
				rt = (r - re) / (rf - re);
				ir = 1.0f;
				ig = 0.0f + 1.0f * rt;
				ib = 0.0f;
			}
			if (rf < r && r <= 1.0f - 0.00001f)
			{
				rt = (r - rf) / (1.0f - rf);
				ir = 1.0f;
				ig = 1.0f;
				ib = 0.0f + 1.0f * rt;
			}
			if (1.0f - 0.00001f < r)
			{
				ir = 1.0f;
				ig = 1.0f;
				ib = 1.0f;
			}

			vec3f emissiveColor(ir / 1.01f, ig / 1.01f, ib / 1.01f);
			return emissiveColor;
		}

		static inline __device__ vec3f voltage_mapping_deriv(float deriv, float maxVol, float minVol, float excitation, float firing)
		{
			float maxDeriv = maxVol - minVol;
			float r = deriv / maxDeriv;
			float ir, ig, ib;
			
			if (r <= 0.00001f)
			{
				ir = 0.1f;
				ig = 0.1f;
				ib = 0.1f;
			}
			if (0.00001f < r && r <= excitation)
			{
				ir = 0.1f + 0.9f * r / excitation;
				ig = 0.1f;
				ib = 0.1f;
			}
			if (excitation < r && r <= firing)
			{
				ir = 1.0f;
				ig = 0.1f + 0.9f * (r - excitation) / (firing - excitation);
				ib = 0.1f;
			}
			if (firing < r && r <= 1.0f - 0.00001f)
			{
				ir = 1.0f;
				ig = 1.0f;
				ib = 0.1f + 0.9f * (r - firing) / (1.0f - firing);
			}
			if (1.0f - 0.00001f < r)
			{
				ir = 1.0f;
				ig = 1.0f;
				ib = 1.0f;
			}

			vec3f emissiveColor(ir / 1.01f, ig / 1.01f, ib / 1.01f);
			return emissiveColor;
		}

		//                                                                      // 
		//************************ BasicNeurons ********************************//
		//                                                                      // 
		OPTIX_INTERSECT_PROGRAM(BasicNeurons)()
		{
			intersectProgram(optixGetPrimitiveIndex());
		}

		// Cylinder boundingBox    
		OPTIX_BOUNDS_PROGRAM(BasicNeurons)(const void *geomData, box3f &primBounds, const int primID)
		{
			const TubesGeom &self = *(const TubesGeom*)geomData;

			//vec3f pa = self.vertices[primID * 2];
			//vec3f ra = self.endPointRadii[primID * 2];
			//primBounds = box3f().including(pa - ra)//self.radius)
			//	.including(pa + ra);//self.radius);

			//vec3f pb = self.vertices[primID * 2 + 1];
			//vec3f rb = self.endPointRadii[primID * 2 + 1];
			//primBounds
			//	= primBounds
			//	.including(pb - rb)//self.radius)
			//	.including(pb + rb);//self.radius);

			vec3f pa, pb;
			float ra, rb;

			primToTube(self, primID, pa, pb, ra, rb);

			primBounds = box3f().including(pa - ra).including(pa + ra).including(pb - rb).including(pb + rb);
		}

		OPTIX_CLOSEST_HIT_PROGRAM(BasicNeurons)()
		{
			owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
			const auto& self = owl::getProgramData<TubesGeom>();
			auto& prd = owl::getPRD<PerRayData>();

			const FrameState* fs = prd.fs;
			const auto epslon = fs->epslon;

			const auto wo = -ray.direction;
			auto N = prd.Ng;
			N = normalize(N);
			if (dot(N, wo) < epslon)
			{
				prd.done = true;
				return;
			}

			Random& rnd = *prd.rnd;

			const unsigned int prevID = self.prevPrims[prd.primID];
			const unsigned int type = self.types[prd.primID];
			
			float voltage, voltagePre, voltageCurrent, deriv, derivPre, derivCurrent;

			const float relative = prd.relativeTubePosition;

			if (prevID == 0)
			{
				voltage = prd.voltagesColor[prd.primID];
				deriv = prd.voltagesDeriv[prd.primID];
			}
			else
			{
				voltagePre = prd.voltagesColor[self.prevPrims[prd.primID] - 1];
				voltageCurrent = prd.voltagesColor[prd.primID];
				
				voltage = voltagePre * relative + voltageCurrent * (1.0f - relative);

				derivPre = prd.voltagesDeriv[self.prevPrims[prd.primID] - 1];
				derivCurrent = prd.voltagesDeriv[prd.primID];

				deriv = derivPre * relative + derivCurrent * (1.0f - relative);
			}

			float v = (voltage - fs->minVoltage) / (fs->maxVoltage - fs->minVoltage);

			const auto voltage_threshold = fs->voltage_threshold;

			DisneyMaterial material = fs->material;

			ShaderMode shadeMode = (ShaderMode)fs->shadeMode;
			if (shadeMode == ShaderMode::No_Lighting) // always emissive
			{
				if (fs->enable_voltage_deriv)
				{
					prd.radiance += voltage_mapping_deriv(deriv, fs->maxVoltage, fs->minVoltage, fs->excitationDeriv, fs->firingDeriv) * fs->emissive_intensity;
				}
				else
				{
					prd.radiance += voltage_mapping(voltage, fs->maxVoltage, fs->minVoltage, fs->inhibition, fs->resting, fs->excitation, fs->firing) * fs->emissive_intensity;
				}				
				prd.done = true;
			}
			else if (shadeMode == ShaderMode::Direct_Lighting)
			{
				if (prd.depth >= 1)
				{
					prd.done = true;
					return;
				}

				if (v > fs->voltage_threshold)
				{
					vec3f emissive_color;

					if (fs->enable_voltage_deriv)
					{
						emissive_color = voltage_mapping_deriv(deriv, fs->maxVoltage, fs->minVoltage, fs->excitationDeriv, fs->firingDeriv) * fs->emissive_intensity;
					}
					else
					{
						emissive_color = voltage_mapping(voltage, fs->maxVoltage, fs->minVoltage, fs->inhibition, fs->resting, fs->excitation, fs->firing) * fs->emissive_intensity;
					}

					float intensity = 1.0f;

					if (fs->enable_light_decay)
					{
						const auto max_decay_distance = 256.f;
						float a = (max_decay_distance - prd.t) / max_decay_distance;
						float b = a <= 0.f ? 0.f : a * a;
						intensity *= b;
					}

					if (type == 5) emissive_color = fs->synapse_type0_color;
					if (type == 6) emissive_color = fs->synapse_type1_color;
					if (type == 7) emissive_color = fs->synapse_type2_color;

					if (type == 5 || type == 6 || type == 7)
					{
						prd.radiance += emissive_color * dot(N, wo) * intensity;
					}
					else if (fs->enable_emissive)
					{
						prd.radiance += emissive_color * fs->emissive_intensity * dot(N, wo) * intensity;
					}

					// material.base_color = emissive_color;
				}

				vec3f v_x, v_y;
				ortho_basis(v_x, v_y, N);
				float wi_pdf;
				vec3f sampled_wi;
				vec3f albedo = sample_disney_brdf(material, N, wo, v_x, v_y, rnd, sampled_wi, wi_pdf);

				if (wi_pdf < epslon || (albedo.x < epslon && albedo.y < epslon && albedo.z < epslon))
				{
					prd.done = true;
					return;
				}
				prd.attenuation *= (albedo * max(dot(sampled_wi, N), epslon) / wi_pdf);
				prd.wi_sampled = sampled_wi;

			}
			else if (shadeMode == ShaderMode::Full_Lighting)
			{
				if (v > fs->voltage_threshold)
				{
					//auto emissivePre = voltage_mapping(voltagePre, fs->maxVoltage, fs->minVoltage, fs->inhibition, fs->resting, fs->excitation, fs->firing);
					//auto emissiveCurrent = voltage_mapping(voltageCurrent, fs->maxVoltage, fs->minVoltage, fs->inhibition, fs->resting, fs->excitation, fs->firing);
					//auto emissive_color = emissivePre * relative + emissiveCurrent * (1.0f - relative);;
					
					//auto emissive_color = voltage_mapping(voltage, fs->maxVoltage, fs->minVoltage, fs->inhibition, fs->resting, fs->excitation, fs->firing);

					//auto emissive_color = 0.5f * (emissive_color1 + emissive_color2);
					vec3f emissive_color;

					if (fs->enable_voltage_deriv)
					{
						emissive_color = voltage_mapping_deriv(deriv, fs->maxVoltage, fs->minVoltage, fs->excitationDeriv, fs->firingDeriv) * fs->emissive_intensity;
					}
					else
					{
						emissive_color = voltage_mapping(voltage, fs->maxVoltage, fs->minVoltage, fs->inhibition, fs->resting, fs->excitation, fs->firing) * fs->emissive_intensity;
					}

					float intensity = 1.0f;

					if (fs->enable_light_decay)
					{
						const auto max_decay_distance = 256.f;
						float a = (max_decay_distance - prd.t) / max_decay_distance;
						float b = a <= 0.f ? 0.f : a * a;
						intensity *= b;
					}

					if (type == 5) emissive_color = fs->synapse_type0_color;
					if (type == 6) emissive_color = fs->synapse_type1_color;
					if (type == 7) emissive_color = fs->synapse_type2_color;

					if (type == 5 || type == 6 || type == 7)
					{
						prd.radiance += emissive_color * dot(N, wo) * intensity;
					}
					else if (fs->enable_emissive)
					{
						prd.radiance += emissive_color * fs->emissive_intensity * dot(N, wo) * intensity;
					}
					
					// material.base_color = emissive_color;
				}

				vec3f v_x, v_y;
				ortho_basis(v_x, v_y, N);
				float wi_pdf;
				vec3f sampled_wi;
				vec3f albedo = sample_disney_brdf(material, N, wo, v_x, v_y, rnd, sampled_wi, wi_pdf);

				if (wi_pdf < epslon || (albedo.x < epslon && albedo.y < epslon && albedo.z < epslon))
				{
					prd.done = true;
					return;
				}
				prd.attenuation *= albedo * max(dot(sampled_wi, N), epslon) / wi_pdf;
				prd.wi_sampled = sampled_wi;

			}
		}

		//                                                                      // 
		//************************ TriangleMesh ********************************//
		//                                                                      //
		OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
		{
			PerRayData& prd = owl::getPRD<PerRayData>();
			const TrianglesGeomData& self = owl::getProgramData<TrianglesGeomData>();

			const int   primID = optixGetPrimitiveIndex();
			const vec3i index = self.index[primID];

			// compute normal:
			const vec3f& A = self.vertex[index.x];
			const vec3f& B = self.vertex[index.y];
			const vec3f& C = self.vertex[index.z];
			vec3f Ng = normalize(cross(B - A, C - A));

			// const vec3f rayDir = optixGetWorldRayDirection();
			// prd = (.2f + .8f * fabs(dot(rayDir, Ng))) * self.color;

			const float u = optixGetTriangleBarycentrics().x;
			const float v = optixGetTriangleBarycentrics().y;

			const auto wo = -1.0f * (vec3f)optixGetWorldRayDirection();
			if (dot(wo, Ng) < 0.0f) Ng = -1.0f * Ng;

			prd.Ng = Ng;
			prd.hit_point = u * A + v * B + (1 - u - v) * C;

			const FrameState* fs = prd.fs;
			const auto epslon = fs->epslon;
			DisneyMaterial material = fs->material;
			Random& rnd = *prd.rnd;

			vec3f col = (self.color)
				? ((1.f - u - v) * self.color[index.x]
					+ u * self.color[index.y]
					+ v * self.color[index.z])
				: vec3f(-1.0);

			if (fs->enable_mesh_transparency)
			{
				switch (prd.transparentDepth) // currently hard coded
				{
					case 0: material.base_color = vec3f(0.2f, 0.5f, 0.2f); break;
					case 1: material.base_color = vec3f(0.2f, 0.5f, 0.2f); break;
					case 2: material.base_color = vec3f(0.1f, 1.0f, 0.1f); break;
					case 3: material.base_color = vec3f(0.1f, 1.0f, 0.1f); break;
					case 4: material.base_color = vec3f(0.1f, 0.1f, 0.5f); break;
					case 5: material.base_color = vec3f(0.1f, 0.1f, 0.5f); break;
					case 6: material.base_color = vec3f(0.0f, 0.0f, 1.0f); break;
					case 7: material.base_color = vec3f(0.0f, 0.0f, 1.0f); break;
					case 8: material.base_color = vec3f(0.5f, 0.5f, 0.1f); break;
					case 9: material.base_color = vec3f(0.5f, 0.5f, 0.1f); break;
					case 10: material.base_color = vec3f(1.0f, 1.0f, 0.0f); break;
					case 11: material.base_color = vec3f(1.0f, 1.0f, 0.0f); break;
					case 12: material.base_color = vec3f(0.5f, 0.1f, 0.1f); break;
					case 13: material.base_color = vec3f(0.5f, 0.1f, 0.1f); break;
					case 14: material.base_color = vec3f(1.0f, 0.0f, 0.0f); break;
				}
			}
			else
			{
				material.base_color = col;
				//material.base_color = vec3f(0.2f, 0.2f, 0.2f);
			}

			
			vec3f v_x, v_y;
			ortho_basis(v_x, v_y, Ng);
			float wi_pdf;
			vec3f sampled_wi;
			vec3f albedo = sample_disney_brdf(material, Ng, wo, v_x, v_y, rnd, sampled_wi, wi_pdf);

			if (wi_pdf < epslon || (albedo.x < epslon && albedo.y < epslon && albedo.z < epslon))
			{
				prd.done = true;
				return;
			}

			if (fs->enable_mesh_transparency)
			{
				prd.attenuation = (albedo * max(dot(sampled_wi, Ng), epslon) / wi_pdf);
			}
			else
			{
				prd.attenuation *= (albedo * max(dot(sampled_wi, Ng), epslon) / wi_pdf);
			}		

			if (prd.transparentDepth >= fs->max_transparency_depth * 2)
			{
				prd.radiance += prd.attenuation;
				//prd.radiance += material.base_color;
				prd.done = true;
				return;
			}

			float transmit = rnd();
			if (fs->enable_mesh_transparency)
			{
				if (transmit < fs->mesh_transparency)
				{
					prd.wi_sampled = -wo;
					prd.transparentDepth += 1;
					prd.depth -= 1;
				}
				else
				{
					if (prd.transparentDepth % 2 == 0)
					{
						prd.wi_sampled = sampled_wi;
						prd.radiance += prd.attenuation;
						//prd.radiance += material.base_color;
						prd.done = true;
						return;
					}
					if (prd.transparentDepth % 2 == 1)
					{
						prd.wi_sampled = -wo;
						prd.transparentDepth += 1;
						prd.depth -= 1;
					}
				}				
			}
			else
			{
				prd.wi_sampled = sampled_wi;
			}
			
		}

		//                                                                      // 
		//************************ Curves ***************************************//
		//                                                                      // 
		inline __device__
			void  getHermiteBoundingBox(const vec3f coA, const vec3f coB, const vec3f coC, const vec3f coD, vec3f& re0, vec3f& re1)
		{
			// derivative of cubic function
			vec3f a = 3.0f * coA;
			vec3f b = 2.0f * coB;
			vec3f c = coC;

			vec3f part = 4.0f * b * b - 12.0f * a * c;
			vec3f partSolution = vec3f(sqrtf(part.x), sqrtf(part.y), sqrtf(part.z));
			// todo : if a equals zero , add small value 
			re0 = (-2.0f * b + partSolution) / (6.0f * a);
			re1 = (-2.0f * b - partSolution) / (6.0f * a);

		}

		OPTIX_INTERSECT_PROGRAM(Curves)()
		{
			intersectCurveProgram(optixGetPrimitiveIndex());
		}

		// Cylinder boundingBox    
		OPTIX_BOUNDS_PROGRAM(Curves)(const void  *geomData, box3f       &primBounds, const int    primID)
		{
			const CurveGeom &self = *(const CurveGeom*)geomData;

			const vec3f coA = self.coefficients[primID * 4];
			const vec3f coB = self.coefficients[primID * 4 + 1];
			const vec3f coC = self.coefficients[primID * 4 + 2];
			const vec3f coD = self.coefficients[primID * 4 + 3];

			const vec4f rad = self.radius[primID];

			vec3f pa = Hermite(0.0f, coA, coB, coC, coD);
			float ra = cubicPolynomial(0.0f, rad);

			vec3f pb = Hermite(1.0f, coA, coB, coC, coD);
			float rb = cubicPolynomial(1.0f, rad);

			vec3f re0, re1;
			getHermiteBoundingBox(coA, coB, coC, coD, re0, re1);

			// add end point to bounding box
			primBounds = box3f().including(pa - ra).including(pa + ra);
			primBounds = primBounds.including(pb - rb).including(pb + rb);

			if (re0.x > 0 && re0.x < 1)
			{
				float t = re0.x;
				vec3f p = Hermite(t, coA, coB, coC, coD);
				primBounds = primBounds.including(p - rb).including(p + rb);
			}
			if (re0.y > 0 && re0.y < 1)
			{
				float t = re0.y;
				vec3f p = Hermite(t, coA, coB, coC, coD);
				primBounds = primBounds.including(p - rb).including(p + rb);
			}
			if (re0.z > 0 && re0.z < 1)
			{
				float t = re0.z;
				vec3f p = Hermite(t, coA, coB, coC, coD);
				primBounds = primBounds.including(p - rb).including(p + rb);
			}

			if (re1.x > 0 && re1.x < 1)
			{
				float t = re1.x;
				vec3f p = Hermite(t, coA, coB, coC, coD);
				primBounds = primBounds.including(p - rb).including(p + rb);
			}
			if (re1.y > 0 && re1.y < 1)
			{
				float t = re1.y;
				vec3f p = Hermite(t, coA, coB, coC, coD);
				primBounds = primBounds.including(p - rb).including(p + rb);
			}
			if (re1.z > 0 && re1.z < 1)
			{
				float t = re1.z;
				vec3f p = Hermite(t, coA, coB, coC, coD);
				primBounds = primBounds.including(p - rb).including(p + rb);
			}

		}

		OPTIX_CLOSEST_HIT_PROGRAM(Curves)()
		{
			const auto& self = owl::getProgramData<CurveGeom>();
			auto& prd = owl::getPRD<PerRayData>();
			const int primID = optixGetPrimitiveIndex();
			owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

			const FrameState* fs = prd.fs;
			const auto epslon = fs->epslon;

			const auto wo = -ray.direction;
			auto Ng = prd.Ng;
			Ng = normalize(Ng);
			if (dot(Ng, wo) < epslon)
			{
				prd.done = true;
				return;
			}

			Random& rnd = *prd.rnd;

			DisneyMaterial material = fs->material;
			//const auto & base_color = material.base_color;
			material.base_color = self.curveColor[primID];

			vec3f v_x, v_y;
			ortho_basis(v_x, v_y, Ng);
			float wi_pdf;
			vec3f sampled_wi;
			vec3f albedo = sample_disney_brdf(material, Ng, wo, v_x, v_y, rnd, sampled_wi, wi_pdf);

			if (wi_pdf < epslon || (albedo.x < epslon && albedo.y < epslon && albedo.z < epslon))
			{
				prd.done = true;
				return;
			}

			prd.attenuation *= (albedo * max(dot(sampled_wi, Ng), epslon) / wi_pdf);
			prd.wi_sampled = sampled_wi;

		}



		//                                                                      // 
		//************************ Quad Light **********************************//
		//                                                                      // 
		static inline __device__
			bool in_quad_light_shadow(OptixTraversableHandle world, const FrameState* fs,
				const vec3f& hit_point, vec3f& light_normal, vec3f &shadow_ray_direction, Random& rnd)
		{
			float r1 = rnd();
			float r2 = rnd();
			const vec3f light_origin = fs->quad_light.p;
			const vec3f light_u = fs->quad_light.v1 - light_origin;
			const vec3f light_v = fs->quad_light.v2 - light_origin;
			const vec3f light_p = light_origin + r1 * light_u * .1f + r2 * light_v * .1f;
			light_normal = normalize(cross(light_u, light_v)); //< important
			shadow_ray_direction = (light_p - hit_point);

			const auto shadow_ray_distance = length(shadow_ray_direction);

			Ray shadow_ray = owl::Ray(hit_point, normalize(shadow_ray_direction), 1e-3f, shadow_ray_distance - 1e-3f);
			PerRayData prd_shadow;
			prd_shadow.primID = -1;
			owl::traceRay(world, shadow_ray, prd_shadow, OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
			if (prd_shadow.t < shadow_ray_distance - fs->epslon) // hit something, in shadow
			{
				return true;
			}
			return false;
		}

		OPTIX_INTERSECT_PROGRAM(QuadLight)()
		{
			owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

			const auto& geo = owl::getProgramData<QuadLightGeom>();
			auto& prd = owl::getPRD<PerRayData>();

			auto anchor = geo.p;
			auto v1 = geo.v1;
			auto v2 = geo.v2;
			vec3f u = v1 - anchor;
			vec3f v = v2 - anchor;
			float u_length = length(u);
			float v_length = length(v);
			vec3f n = normalize(cross(u, v));
			float dt = dot(ray.direction, n);
			float t = (dot(n, anchor) - dot(n, ray.origin)) / dt;
			if (t > ray.tmin && t < ray.tmax)
			{
				vec3f p = ray.origin + ray.direction * t;
				vec3f vi = p - anchor;
				float a1 = dot(u, vi) / u_length;
				float a2 = dot(v, vi) / v_length;
				if (a1 >= 0 && a1 <= u_length && a2 >= 0 && a2 <= v_length)
				{
					if (optixReportIntersection(t, 0))
					{
						//prd.primID = 0;
						prd.meshID = 1;
						prd.t = t;
						prd.Ng = n;
					}
				}
			}
		}

		OPTIX_BOUNDS_PROGRAM(QuadLight)(const void  *geomData, box3f &primBounds, const int primID)
		{
			const QuadLightGeom &self = *(const QuadLightGeom*)geomData;
			auto anchor = self.p;
			auto v1 = self.v1;
			auto v2 = self.v2;
			const auto tv1 = v1 - anchor;
			const auto tv2 = v2 - anchor;
			const auto p00 = anchor;
			const auto p01 = anchor + tv1;
			const auto p10 = anchor + tv2;
			const auto p11 = anchor + tv1 + tv2;

			primBounds = box3f();
			primBounds.extend(p00);
			primBounds.extend(p01);
			primBounds.extend(p10);
			primBounds.extend(p11);
		}

		OPTIX_CLOSEST_HIT_PROGRAM(QuadLight)()
		{
			//const auto& light = owl::getProgramData<QuadLightGeom>();
			auto& prd = owl::getPRD<PerRayData>();
			const auto epslon = prd.fs->epslon;
			const auto max_decay_distance = prd.fs->max_decay_distance;
			const auto light_decay_index = prd.fs->light_decay_index;
			const auto light_decay_enable = prd.fs->enable_light_decay;
			const auto color = prd.fs->quad_light.color;
			const auto ray_distance = prd.t;
			auto intensity = prd.fs->quad_light.intensity;

			if (prd.depth > 0 && light_decay_enable)
			{
				float a = (max_decay_distance - ray_distance) / max_decay_distance;
				float b = a <= epslon ? epslon : pow(a, light_decay_index);
				intensity *= b;
			}
			prd.radiance += (color * intensity) * prd.attenuation;
			prd.done = true;
		}

		//                                                                      // 
		//************************ Point Light *********************************//
		//                                                                      // 
		static inline __device__
			bool in_point_light_shadow(OptixTraversableHandle world, const FrameState* fs,
				const vec3f& hit_point, vec3f &shadow_ray_direction, Random& rnd)
		{
			shadow_ray_direction = (fs->point_light_pos - hit_point);
			const float shadow_ray_distance = length(shadow_ray_direction);

			Ray shadow_ray = owl::Ray(hit_point, normalize(shadow_ray_direction), 1e-3f, shadow_ray_distance - 1e-3f);
			PerRayData prd_shadow;
			prd_shadow.primID = -1;
			owl::traceRay(world, shadow_ray, prd_shadow, OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
			if (prd_shadow.t < shadow_ray_distance - fs->epslon) // hit something, in shadow
			{
				return true;
			}
			return false;
		}

		//                                                                      // 
		//************************ Ground **************************************//
		//                                                                      // 
		OPTIX_INTERSECT_PROGRAM(Ground)()
		{
			owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

			const auto& self = owl::getProgramData<QuadGeom>();
			auto& prd = owl::getPRD<PerRayData>();

			auto anchor = self.p;
			auto v1 = self.v1;
			auto v2 = self.v2;
			vec3f u = v1 - anchor;
			vec3f v = v2 - anchor;
			float u_length = length(u);
			float v_length = length(v);
			vec3f n = normalize(cross(u, v));
			float dt = dot(ray.direction, n);
			float t = (dot(n, anchor) - dot(n, ray.origin)) / dt;
			if (t > ray.tmin && t < ray.tmax)
			{
				vec3f p = ray.origin + ray.direction * t;
				vec3f vi = p - anchor;
				float a1 = dot(u, vi) / u_length;
				float a2 = dot(v, vi) / v_length;
				if (a1 >= 0 && a1 <= u_length && a2 >= 0 && a2 <= v_length)
				{
					if (optixReportIntersection(t, 0))
					{
						//prd.primID = 0;
						prd.meshID = 1;
						prd.t = t;
						prd.Ng = n;
					}
				}
			}
		}

		OPTIX_BOUNDS_PROGRAM(Ground)(const void  *geomData, box3f &primBounds, const int primID)
		{
			const QuadGeom &self = *(const QuadGeom*)geomData;
			auto anchor = self.p;
			auto v1 = self.v1;
			auto v2 = self.v2;
			const auto tv1 = v1 - anchor;
			const auto tv2 = v2 - anchor;
			const auto p00 = anchor;
			const auto p01 = anchor + tv1;
			const auto p10 = anchor + tv2;
			const auto p11 = anchor + tv1 + tv2;

			primBounds = box3f();
			primBounds.extend(p00);
			primBounds.extend(p01);
			primBounds.extend(p10);
			primBounds.extend(p11);
		}

		OPTIX_CLOSEST_HIT_PROGRAM(Ground)()
		{
			owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());
			auto& prd = owl::getPRD<PerRayData>();

			Random& rnd = *prd.rnd;
			const vec3f albedo = prd.fs->ground_color;
			const vec3f hit_point = ray.origin + prd.t * ray.direction;
			prd.hit_point = hit_point;

			const vec3f N = prd.Ng;
			vec3f wi; //< sampled direction

			Onb onb(N);
			float r1 = rnd();
			float r2 = rnd();
			cosine_sample_hemisphere(r1, r2, wi);
			onb.inverse_transform(wi);

			prd.attenuation *= albedo;
			prd.wi_sampled = wi;

			vec3f shadow_ray_dir;
			vec3f light_normal;

			const auto epslon = prd.fs->epslon;
			const auto max_decay_distance = prd.fs->max_decay_distance;
			const auto light_decay_index = prd.fs->light_decay_index;
			const auto light_decay_enable = prd.fs->enable_light_decay;

			if (prd.depth == 0)
			{
				auto enable_quad_light = prd.fs->enable_quad_light;
				if (enable_quad_light && !in_quad_light_shadow(prd.world, prd.fs, hit_point, light_normal, shadow_ray_dir, rnd))
				{
					float NdotL_hit_point = dot(N, shadow_ray_dir);
					float NdotL_light = dot(light_normal, -shadow_ray_dir);
					if (NdotL_light > epslon && NdotL_hit_point > epslon)
					{
						auto intensity = prd.fs->point_light_intensity;
						const auto shadow_ray_distance = length(shadow_ray_dir);
						if (light_decay_enable)
						{
							float a = (max_decay_distance - shadow_ray_distance) / max_decay_distance;
							float b = a <= 0.f ? 0.f : pow(a, light_decay_index);
							intensity *= b;
						}
						prd.radiance += intensity * albedo;
					}
				}

				auto enable_point_light = prd.fs->enable_point_light;
				if (enable_point_light && !in_point_light_shadow(prd.world, prd.fs, hit_point, shadow_ray_dir, rnd))
				{
					float NdotL_hit_point = dot(N, shadow_ray_dir);
					if (NdotL_hit_point > epslon)
					{
						auto intensity = prd.fs->point_light_intensity;
						const auto shadow_ray_distance = length(shadow_ray_dir);
						if (light_decay_enable)
						{
							float a = (max_decay_distance - shadow_ray_distance) / max_decay_distance;
							float b = a <= 0.f ? 0.f : pow(a, light_decay_index);
							intensity *= b;
						}
						prd.radiance += intensity * albedo;
					}
				}
			}

		}

	}

}
