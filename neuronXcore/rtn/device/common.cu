#include "hip/hip_runtime.h"
﻿// ======================================================================== //
// Copyright 2018-2020 The Contributors                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "rtn/device/TubesGeom.h"
#include "rtn/device/PerRayData.h"
#include "rtn/device/RayGenData.h"
#include "rtn/device/Camera.h"
#include "rtn/device/disney_bsdf.h"

#include <optix_device.h> // Only for test, can be remove later. 

#define HIGH -24.0f
#define MID1 -42.0f
#define MID2 -60.0f
#define LOW -100.0f
#define DELTA 18.0f

namespace rtn {
	namespace device {

		__device__ inline
			vec4f ToneMap(const vec4f& c, float limit)
		{
			float luminance = 0.3f * c.x + 0.6f * c.y + 0.1f * c.z;

			vec4f col = c * 1.0f / (1.0f + luminance / limit);
			return vec4f(col.x, col.y, col.z, 1.0f);
		}

		__device__ inline
			vec4f LinearToSRGB(const vec4f& c)
		{
			const float kInvGamma = 1.0f / 2.2f;
			return vec4f(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma), c.w);
		}


		static inline __device__
			int32_t make_8bit(const float f)
		{
			return min(255, max(0, int(f * 256.f)));
		}

		static     inline __device__
			int32_t make_rgba8(const vec4f color)
		{
			return
				(make_8bit(color.x) << 0) +
				(make_8bit(color.y) << 8) +
				(make_8bit(color.z) << 16) + 
				(make_8bit(color.w) << 24);
		}

		inline __device__ vec3f random_in_unit_sphere(Random& rnd) {
			vec3f p;
			do {
				p = 2.0f * vec3f(rnd(), rnd(), rnd()) - vec3f(1.f, 1.f, 1.f);
			} while (dot(p, p) >= 1.0f);
			return p;
		}

		inline __device__  vec3f toneMapping(const vec3f& color)
		{
			float a = 2.51f;
			float b = 0.03f;
			float c = 2.43f;
			float d = 0.59f;
			float e = 0.14f;
			return saturate((color*(a*color + b)) / (color*(color*color + d) + e));
		}


		inline __device__
			vec3f vmin(vec3f x, vec3f y)
		{
			vec3f res = vec3f(0.0f);
			res.x = min(x.x, y.x);
			res.y = min(x.y, y.y);
			res.z = min(x.z, y.z);
			return res;
		}


		inline __device__
			vec3f vmax(vec3f x, vec3f y)
		{
			vec3f res = vec3f(0.0f);
			res.x = max(x.x, y.x);
			res.y = max(x.y, y.y);
			res.z = max(x.z, y.z);
			return res;
		}

		inline __device__
			vec3f vpow(vec3f x, float y)
		{
			vec3f res = vec3f(0.0f);
			res.x = pow(x.x, y);
			res.y = pow(x.y, y);
			res.z = pow(x.z, y);
			return res;
		}

		inline __device__
			vec3f linear2srgb(vec3f col)
		{
			col = vmax(6.10352e-5, col);
			return vmin(col * 12.92f, vpow(vmax(col, 0.00313067), 1.0 / 2.4) * 1.055f - 0.055f);
		}


		//inline __device__
		//	vec3f vpow(vec3f x, float y)
		//{
		//	vec3f res = vec3f(0.0f);
		//	res.x = pow(x.x, y);
		//	res.y = pow(x.y, y);
		//	res.z = pow(x.z, y);
		//	return res;
		//}

		// inline __device__  vec3f linear2srgb(const vec3f& color)
		//{
		//	return vpow(color, 1.f / 2.2f);
		//}


		inline __device__
			void pathTrace(const RayGenData& self, owl::Ray& ray, Random& rnd, PerRayData& prd)
		{
			prd.attenuation = vec3f(1.0f);
			prd.radiance = vec3f(0.0f);
			prd.depth = 0;
			prd.transparentDepth = 0;
			prd.alpha = 0.0f;
			prd.fs = &self.frameStateBuffer[0];
			prd.voltagesColor = self.voltagesColor;
			prd.voltagesDeriv = self.voltagesDeriv;
			prd.rnd = &rnd;
			//prd.voltageColorMap = self.voltageColorMap;
			prd.done = false;
			prd.world = self.world;
			const int pathDepth = prd.fs->pathDepth;
			for (int depth = 0; depth < pathDepth; depth++) /* iterative version of recursion */
			{
				owl::traceRay(self.world, ray, prd, OPTIX_RAY_FLAG_DISABLE_ANYHIT);
				if (prd.done == true) 
				{
					//if (depth > 0) 
					break;
				}
				
				prd.depth++;
				depth = prd.depth;
				ray = owl::Ray(prd.hit_point, normalize(prd.wi_sampled), 1e-3f, 1e6f);
			}
		}

		/*! the actual ray generation program - note this has no formal
			function parameters, but gets its paramters throught the 'pixelID'
			and 'pixelBuffer' variables/buffers declared above */
		OPTIX_RAYGEN_PROGRAM(raygen_program)()
		{
			const RayGenData& self = owl::getProgramData<RayGenData>();
			const vec2i pixelID = owl::getLaunchIndex();
			const vec2i launchDim = owl::getLaunchDims();

			if (pixelID.x >= self.fbSize.x) return;
			if (pixelID.y >= self.fbSize.y) return;
			const int pixelIdx = pixelID.x + self.fbSize.x * pixelID.y;

			// for multi-gpu check 
			if (((pixelID.x >> 5) % self.deviceCount) != self.deviceIndex) return;

			const FrameState* fs = &self.frameStateBuffer[0];
			const int accumID = fs->accumID;
			const int spp = fs->samplesPerPixel;
			int pixel_index = pixelID.y * launchDim.x + pixelID.x;
			Random rnd(pixel_index, accumID);

			vec4f col(0.f);
			PerRayData prd;
			for (int s = 0; s < spp; s++)
			{
				vec2f pixelSample = vec2f(pixelID) + vec2f(rnd(), rnd());
				owl::Ray ray = Camera::generateRay(*fs, pixelSample, rnd);
				pathTrace(self, ray, rnd, prd);
				col += vec4f(prd.radiance, prd.alpha);
			}
			col = col / float(spp);

			const int maxAccum = fs->maxAccum;

			vec4f perFrameColor;

			if (maxAccum <= 0)
			{
				if (accumID > 0)
				{
					col = col + (vec4f)self.accumBufferPtr[pixelIdx];
				}
				self.accumBufferPtr[pixelIdx] = col;
				perFrameColor = col / (accumID + 1.f);
			}
			else /* smooth color change for situations with fixed camera */
			{
				const int accumStatus = min(accumID, maxAccum);
				if (accumID == 0)
				{
					self.accumBufferPtr[pixelIdx] = col;
				}
				if (accumID > 0 && accumID < maxAccum)
				{
					col = col + (float)accumID * (vec4f)self.accumBufferPtr[pixelIdx];
					self.accumBufferPtr[pixelIdx] = col / (accumID + 1.f);
				}
				if (accumID >= maxAccum)
				{
					col = col + (float)maxAccum * (vec4f)self.accumBufferPtr[pixelIdx];
					self.accumBufferPtr[pixelIdx] = col / ((float)maxAccum + 1.f);
				}
				perFrameColor = col / (accumStatus + 1.f);
			}

			vec3f hdrColor = vec3f(perFrameColor.x, perFrameColor.y, perFrameColor.z);
			vec3f ldrColor = toneMapping(hdrColor);
			vec3f sRGBColor = linear2srgb(ldrColor);

			uint32_t rgba = make_rgba8(vec4f(sRGBColor, perFrameColor.w));
			//uint32_t rgba = make_rgba8(col);

			self.colorBufferPtr[pixelIdx] = rgba;
		}

		OPTIX_MISS_PROGRAM(miss_program)()
		{
			const auto& miss_data = owl::getProgramData<MissData>();
			const vec3f top = miss_data.topColor;
			const vec3f bottom = miss_data.bottomColor;
			const vec3f ray_dir = optixGetWorldRayDirection();
			const float t = ray_dir.y * 0.5f + 0.5f; // Transform the y component from [-1.0f, 1.0f] to [0.0f, 1.0f]
			const vec3f c = lerp(top, bottom, t);

			PerRayData& prd = owl::getPRD<PerRayData>();
			prd.done = true;
			if (prd.depth == 0)
			{
				prd.radiance += c;
				prd.alpha = 0.f;
			}
			else
			{
				//prd.radiance += prd.attenuation * c * 1.618f; 
				prd.radiance += prd.attenuation * c * 0.618f;
				prd.alpha = 1.0f;
			}
		}

		OPTIX_MISS_PROGRAM(miss_program_shadow)()
		{ }
	}
}

